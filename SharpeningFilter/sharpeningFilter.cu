#include "hip/hip_runtime.h"
//
// CUDA implementation of Image Sharpening Filter
//
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

// Run Sharpening Filter on GPU
__global__ void sharpeningFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   float kernel[FILTER_WIDTH][FILTER_HEIGHT] = {-1, -1, -1, -1, 9, -1, -1, -1, -1};
   // only threads inside image will write results
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
      for(int c=0 ; c<channel ; c++)   
      {
         // Sum of pixel values 
         float sum = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))*channel+c];
               sum += fl*kernel[ky+FILTER_HEIGHT/2][kx+FILTER_WIDTH/2];
            }
         }
         dstImage[(y*width+x)*channel+c] =  sum;
      }
   }
}


// The wrapper is used to call sharpening filter 
extern "C" void sharpeningFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of image channels
        int channel = input.step/input.cols; 

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        unsigned char *d_input, *d_output;
        
        // Allocate device memory
        hipMalloc<unsigned char>(&d_input,inputSize);
        hipMalloc<unsigned char>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        sharpeningFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, channel);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}













