#include "filter_dispatch.h"
#include "filter_common.cuh"

// External filter functions
extern void applyPrewitt(hipSurfaceObject_t, int, int);
extern void applyBlur(hipSurfaceObject_t, int, int);
extern void applyLaplacian(hipSurfaceObject_t, int, int);
extern void applyGrayscaleInvert(hipSurfaceObject_t, int, int);
extern void applySharpen(hipSurfaceObject_t, int, int);
extern void applySepia(hipSurfaceObject_t, int, int);
extern void applyMean(hipSurfaceObject_t, int, int);
extern void applyEmboss(hipSurfaceObject_t, int, int);
//...

void applyFilter(FilterType type, hipArray_t destArray, int width, int height) {
    hipResourceDesc desc = {};
    desc.resType = hipResourceTypeArray;
    desc.res.array.array = destArray;
    
    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &desc);

    switch (type) {
        enum class FilterType {
            None,
            Sobel,
            Prewitt,
            Blur,
            Laplacian,
            GrayscaleInvert,
            Invert,
            Sharpen,
            Sepia,
            Mean,
            Emboss
        };

    hipDestroySurfaceObject(surface);
}
