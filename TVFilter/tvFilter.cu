#include "hip/hip_runtime.h"
//
// CUDA implementation of Total Variation Filter
//
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE      16
#define FILTER_WIDTH    3       
#define FILTER_HEIGHT   3       

using namespace std;

// Run Total Variation Filter on GPU
__global__ void tvFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   // only threads inside image will write results
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
         float sod = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))];
               float center = srcImage[((y)*width + (x))];
               sod += fl-center;
            }
         }
         dstImage[(y*width+x)] = sod;
   }
}


// The wrapper is used to call total variation filter 
extern "C" void tvFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Calculate number of input & output bytes in each block
        const int inputSize = input.cols * input.rows;
        const int outputSize = output.cols * output.rows;
        unsigned char *d_input, *d_output;

        // Allocate device memory
        hipMalloc<unsigned char>(&d_input,inputSize);
        hipMalloc<unsigned char>(&d_output,outputSize);

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start);

        // Run BoxFilter kernel on CUDA 
        tvFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows);

        // Stop time
        hipEventRecord(stop);

        //Copy data from device memory to output image
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);

        //Free the device memory
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        
        // Calculate elapsed time in milisecond  
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nTotal processing time on GPU (ms): " << milliseconds << "\n";
}













